#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <stdio.h>
#include <cmath>

#include "hip/hip_runtime.h"
#include ""
#include <thrust\host_vector.h>
#include <thrust\device_vector.h>

using namespace std;

template <class Type>
class LinearRegression
{
private:
    int row, col;
    Type** trainX;
    Type* trainY;
    Type* theta;
    float learnRate;
    float precision;
public:
    //__host__ __device__ LinearRegression() { trainX = { {} }; trainY = {}; learnRate = 0.0; precision = 0.0; }
    LinearRegression(vector<vector<Type> > & featureX, vector<Type> & featureY, float convergenceRate, float convergencePrecision);
    __host__ __device__ int Row() const { return row; };
    __host__ __device__ int Col() const { return col; };
    __host__ __device__ Type hypothesis(Type* x);
    __host__ __device__ Type costFunction();
    __host__ __device__ Type partialDerivation(int index);
    __host__ __device__ void updateTheta();
    __host__ __device__ void train();
    ~LinearRegression();
};

template<class Type>
LinearRegression<Type>::LinearRegression(vector<vector<Type> > & featureX, vector<Type> & featureY, float convergenceRate, float convergencePrecision)
{
    row = featureX.size(); col = featureX[0].size();

    // initialize trainX
    trainX = new Type*[col];
    for (size_t i = 0; i < col; i++)
    {
        trainX[i] = new Type[row];
    }
    for (size_t i = 0; i < row; i++)
    {
        for (size_t j = 0; j < col; j++)
        {
            trainX[i][j] = featureX[i][j];
        }
    }

    trainY = new Type[row];
    for (size_t i = 0; i < row; i++)
    {
        trainY[i] = featureY[i];
    }

    theta = new Type[col];
    for (size_t i = 0; i < col; i++)
    {
        theta[i] = (Type)0;
    }
    
    learnRate = convergenceRate;
    precision = convergencePrecision;
}

template <class Type>
__host__ __device__ Type LinearRegression<Type>::hypothesis(Type* x)
{
    Type hypo = 0;
    for (size_t i = 0; i < col; i++)
    {
        hypo += x[i] * theta[i];
    }
    return hypo;
}

template <class Type>
__host__ __device__ Type LinearRegression<Type>::costFunction()
{
    Type cost = 0;
    for (size_t i = 0; i < row; i++)
    {
        Type t = hypothesis(trainX[i]) - trainY[i];
        cost += powf(t, 2);
    }
    return 0.5*cost / row;
}

template <class Type>
__host__ __device__ Type LinearRegression<Type>::partialDerivation(int index)
{
    Type partial = 0;
    for (size_t i = 0; i < row; i++)
    {
        Type t = hypothesis(trainX[i]) - trainY[i];
        partial += t*trainX[i][index];
    }
    return partial / row;
}

template <class Type>
__host__ __device__ void LinearRegression<Type>::updateTheta()
{
    for (size_t i = 0; i < col; i++)
    {
        theta[i] -= learnRate*partialDerivation(i);
    }
}

template <class Type>
__host__ __device__ void LinearRegression<Type>::train()
{
    Type lastCostValue = costFunction();
    int epoch = 0;
    for (;;)
    {
        epoch += 1;
        // update theta
        updateTheta();
        if (fabsf(costFunction() - lastCostValue) <= precision)break;
        else lastCostValue = costFunction();
    }
    printf("%f, %f\n", theta[0], theta[1]);
}

template <class Type>
__host__ __device__ LinearRegression<Type>::~LinearRegression()
{
    // delete trainX
    for (size_t i = 0; i < row; i++)
    {
        delete[] trainX[i];
    }
    delete[] trainX;
    // deelte trainY
    delete[] trainY;
    // delete theta
    delete[] theta;
}

int main()
{
    vector<vector<float> > x = { {1,1},{1,2},{1,3} };
    vector<float> y = { 6,10,14 };
    float lr = 0.3;
    float error = 1e-4;

    LinearRegression<float> ln(x, y, lr, error);
    ln.train();

    system("pause");
    return 0;
}