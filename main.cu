#include <iostream>
#include <vector>
#include <stdio.h>
#include <cmath>

#include "hip/hip_runtime.h"
#include ""
#include <thrust\host_vector.h>
#include <thrust\device_vector.h>

using namespace std;

template <class Type>
class Array
{
private:
    Type* array;
    unsigned int size;
public:
    Array() {};
    Array(const int length);
    int Size() const { return size; };
    void changeValue(int index, Type newValue);
    ~Array() { delete[] array; };
};

template <class Type>
Array<Type>::Array(const int length)
{
    size = length;
    array = new Type[lengt];
    for (size_t i = 0; i < size; i++)
    {
        a[i] = rand() % 10;
    }
}

template <class Type>
void Array<Type>::changeValue(int index, Type newValue)
{
    if (index > size)
    {
        cout << "out of range" << endl;
        return;
    }
    a[index] = newValue;
}

template <class Type>
class Matrix
{
private:
    Type** matrix;
    int row, col;
public:
    //Matrix();
    Matrix(const int r, const int c);
    void changeValue(int i, int j, Type newValue);
    int Row() const { return row; };
    int Col() const { return col; };
    ~Matrix();
};

template <class Type>
Matrix<Type>::Matrix(const int r, const int c)
{
    row = r, col = c;
    matrix = new Type *[col];
    for (size_t i = 0; i < col; i++)
    {
        matrix[i] = new Type[row];
    }

    for (size_t i = 0; i < row; i++)
    {
        for (size_t j = 0; j < col; j++)
        {
            matrix[i][j] = rand() % 10;
        }
    }
}

template <class Type>
void Matrix<Type>::changeValue(int i, int j, Type newValue)
{
    if (i > row || j > col)
    {
        cout << "out of range" << endl;
        return;
    }
    matrix[i][j] = newValue;
}

template <class Type>
Matrix<Type>::~Matrix()
{
    for (size_t i = 0; i < col; i++)
    {
        delete[] matrix[i];
    }
    delete[] matrix;
}

template <class Type>
class LinearRegression
{
private:
    vector<vector<Type> > trainX;
    vector<Type> trainY;
    Type learnRate;
    Type precision;
public:
    LinearRegression() { trainX = { {} }; trainY = {}; learnRate = (Type)0; precision = (Type)0; }
    LinearRegression(vector<vector<Type> > & featureX, vector<Type> & featureY, Type convergenceRate, Type convergencePrecision);
    __host__ __device__ Type hypothesis();
    __host__ __device__ Type costFunction();
    __host__ __device__ Type partialDerivation();
    __host__ __device__ void train();
    ~LinearRegression() {};
};

template<class Type>
LinearRegression<Type>::LinearRegression(vector<vector<Type> > & featureX, vector<Type> & featureY, Type convergenceRate, Type convergencePrecision)
{
    trainX = featureX;
    trainY = featureY;
    learnRate = convergenceRate;
    precision = convergencePrecision;
}

template <class Type>
__host__ __device__ Type LinearRegression<Type>::hypothesis()
{

}

template <class Type>
__host__ __device__ Type LinearRegression<Type>::costFunction()
{

}

int main()
{
    const int N = 3;
    vector<int> k = { 1,2,4 };
    thrust::host_vector<int> h_m = k;
    thrust::host_vector<int> h_n = k;

    thrust::device_vector<int> d_m = h_m;
    thrust::device_vector<int> d_n = h_n; 
    thrust::device_vector<int> d_c;

    int f[N] = { h_m.data };


    cout << f[0] << endl;
    cout << f[1] << endl;
    cout << f[2] << endl;
    system("pause");
    return 0;
}